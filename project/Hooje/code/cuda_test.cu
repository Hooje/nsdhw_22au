// cuda_test.cu
#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(){
    printf("inside in kernel\n");
}

int my_cuda(int a, int b){

   kernel<<<1,10>>>();
   hipDeviceSynchronize();

   return 0;
}